#include "hip/hip_runtime.h"
#include <algorithm>

#include "model/decoder.h"
#include "model/encoder.h"
#include "tools/util.h"

/**
@file
Example of how to run transformer inference using our implementation.
*/

// Appoint precision.
#ifdef FP16_MODE
const lightseq::cuda::OperationType OPTYPE =
    lightseq::cuda::OperationType::FP16;
#else
const lightseq::cuda::OperationType OPTYPE =
    lightseq::cuda::OperationType::FP32;
#endif

int main(int argc, char *argv[]) {
  /* ---step1. init environment--- */
  hipStream_t stream_;
  hipblasHandle_t hd_;
  hipSetDevice(0);
  hipStreamCreate(&stream_);
  hipblasCreate(&hd_);
  hipblasSetStream(hd_, stream_);
  typedef lightseq::cuda::OperationTypeTraits<OPTYPE> optraits;

  /* ---step2. load model weights into GPU memory--- */
  lightseq::cuda::TransformerWeight<OPTYPE> tw_;
  // saved in custom proto file
  std::string model_weights_path = argv[1];
  std::string res = tw_.initializing(model_weights_path);
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 0;
  }

  /*
    step3. instantiate encoder and decoder, init the gpu memory buffer.
      using thrust vector to avoid manage gpu memory by hand
  */
  // instantiate encoder
  int max_batch_size = 8;
  thrust::device_vector<int> d_input_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_padding_mask_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_encoder_output_ =
      std::vector<int>(max_batch_size * tw_._max_step * tw_._hidden_size, 0);
  thrust::device_vector<int> d_output_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  std::shared_ptr<lightseq::cuda::Encoder<OPTYPE>> encoder_ =
      std::make_shared<lightseq::cuda::Encoder<OPTYPE>>(
          max_batch_size,
          reinterpret_cast<int *>(thrust::raw_pointer_cast(d_input_.data())),
          reinterpret_cast<int *>(
              thrust::raw_pointer_cast(d_padding_mask_.data())),
          reinterpret_cast<optraits::DataType *>(
              thrust::raw_pointer_cast(d_encoder_output_.data())),
          tw_, stream_, hd_);
  res = encoder_->check();
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 1;
  }
  // instantiate decoder
  std::shared_ptr<lightseq::cuda::Decoder<OPTYPE>> decoder_ =
      std::make_shared<lightseq::cuda::Decoder<OPTYPE>>(
          max_batch_size,
          reinterpret_cast<int *>(
              thrust::raw_pointer_cast(d_padding_mask_.data())),
          reinterpret_cast<optraits::DataType *>(
              thrust::raw_pointer_cast(d_encoder_output_.data())),
          reinterpret_cast<int *>(thrust::raw_pointer_cast(d_output_.data())),
          tw_, stream_, hd_, false,
          reinterpret_cast<int *>(thrust::raw_pointer_cast(d_input_.data())));
  res = decoder_->check();
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 1;
  }
  // init gpu memory buffer
  long buf_bytesize = std::max(encoder_->compute_buffer_bytesize(),
                               decoder_->compute_buffer_bytesize());
  thrust::device_vector<int> d_buf_ =
      std::vector<int>(buf_bytesize / sizeof(int), 0);
  // encoder and decoder use the same buffer to save gpu memory useage
  encoder_->init_buffer(
      reinterpret_cast<void *>(thrust::raw_pointer_cast(d_buf_.data())));
  decoder_->init_buffer(
      reinterpret_cast<void *>(thrust::raw_pointer_cast(d_buf_.data())));
  hipStreamSynchronize(stream_);

  /* ---step4. read input token ids from file--- */
  int batch_size;
  int batch_seq_len;
  std::vector<int> host_input;
  // the first line of input file should
  // be two integers: batch_size and batch_seq_len.
  // followed by batch_size lines of
  // batch_seq_len integers, e.g.
  // 2 3
  // 666 666 666
  // 666 666 666
  std::string input_file_name = argv[2];
  lightseq::cuda::read_batch_tokenids_from_file(input_file_name, batch_size,
                                                batch_seq_len, host_input);

  /* ---step5. infer and log--- */
  int n_tests = 100;
  auto start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < n_tests; i++) {
    // copy inputs from cpu memory to gpu memory
    hipMemcpyAsync(
        reinterpret_cast<int *>(thrust::raw_pointer_cast(d_input_.data())),
        host_input.data(), sizeof(int) * batch_size * batch_seq_len,
        hipMemcpyHostToDevice, stream_);
    encoder_->run_one_infer(batch_size, batch_seq_len);
    decoder_->run_one_infer(batch_size, batch_seq_len);
  }
  auto finish = std::chrono::high_resolution_clock::now();
  auto average_time_consumed = (finish - start) / n_tests;
  std::cout << "time consumed: " << average_time_consumed << std::endl;
  return 0;
}
